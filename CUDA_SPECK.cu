#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdint.h>
#define ROR(x, r) ((x >> r) | (x << (64 - r))) 
#define ROL(x, r) ((x << r) | (x >> (64 - r)))
#define R(x, y, k) (x = ROR(x, 8), x += y, x ^= k, y = ROL(y, 3), y ^= x)
// SPECK64
#define ER32(x,y,k) (x=ROTR32(x,8), x+=y, x^=k, y=ROTL32(y,3), y^=x)
#define ER24(x,y,k) (x=ROTR24(x,8)& 0xffffff, x+=y, x&=0xffffff, x^=k, y=ROTL24(y,3)& 0xffffff, y^=x)
#define ER16(x,y,k) (x=ROTR16(x,7), x+=y, x^=k, y=ROTL16(y,2), y^=x)
#define ER16b(x,y,k) (x=ROTR16b(x,7), x+=y, x^=k, y=ROTL16b(y,2), y^=x)
#define DR32(x,y,k) (y^=x, y=ROTR32(y,3), x^=k, x-=y, x=ROTL32(x,8))
#define ROTL32(x,r) (((x)<<(r)) | (x>>(32-(r))))
#define ROTL24(x,r) (((x)<<(r)) | (x>>(24-(r))))
#define ROTL16(x,r) (((x)<<(r)) | (x>>(16-(r))))
#define ROTL16b(x,r) ((((x)<<(r)) | (x>>(16-(r))))&0xFFFF)
#define ROTR32(x,r) (((x)>>(r)) | ((x)<<(32-(r))))
#define ROTR24(x,r) (((x)>>(r)) | ((x)<<(24-(r))))
#define ROTR16(x,r) (((x)>>(r)) | ((x)<<(16-(r))))
#define ROTR16b(x,r) ((((x)>>(r)) | ((x)<<(16-(r))))&0xFFFF)
#define ROTL64(x,r) (((x)<<(r)) | (x>>(64-(r))))
#define ROTR64(x,r) (((x)>>(r)) | ((x)<<(64-(r))))


#define ROUNDS 32
#define BLOCKS				1024
#define THREADS				1024  // Cannot be less than 256

void Speck6496KeySchedule(uint32_t K[], uint32_t rk[]) {
    uint32_t i, C = K[2], B = K[1], A = K[0];
    for (i = 0; i < 26;) {
        rk[i] = A; ER32(B, A, i++);
        rk[i] = A; ER32(C, A, i++);
    }
}
void Speck6496Encrypt(uint32_t Pt[], uint32_t Ct[], uint32_t rk[]) {
    uint32_t i;
    Ct[0] = Pt[0]; Ct[1] = Pt[1];
    for (i = 0; i < 26;) {
        ER32(Ct[1], Ct[0], rk[i++]);
        printf("plaintext: %08x 0%08x\n", Ct[0], Ct[1]);
    }
}
void Speck6496Encrypt2(uint32_t Pt[], uint32_t Ct[], uint32_t K[]) {
    uint32_t i=0, C = K[2], B = K[1], A = K[0];
    Ct[0] = Pt[0]; Ct[1] = Pt[1];
    
    for (int j = 0; j < 12;j++) {
        ER32(Ct[1], Ct[0], A);
        ER32(B, A, i++);
        ER32(Ct[1], Ct[0], A); 
        ER32(C, A, i++);
    }
    ER32(Ct[1], Ct[0], A);
    ER32(B, A, i++);
    ER32(Ct[1], Ct[0], A);
}
void Speck6472Encrypt2(uint32_t Pt[], uint32_t Ct[], uint32_t K[]) {
    uint32_t i = 0, C = K[2], B = K[1], A = K[0];
    Ct[0] = Pt[0]; Ct[1] = Pt[1];

    for (int j = 0; j < 10; j++) {
        ER24(Ct[1], Ct[0], A);
        ER24(B, A, i++);
        ER24(Ct[1], Ct[0], A);
        ER24(C, A, i++);
    }
    ER24(Ct[1], Ct[0], A);
    ER24(B, A, i++);
    ER24(Ct[1], Ct[0], A);
}
void Speck6464Encrypt(uint16_t Pt[], uint16_t Ct[], uint16_t K[]) {
    uint16_t i = 0, D = K[3], C = K[2], B = K[1], A = K[0];
    Ct[0] = Pt[0]; Ct[1] = Pt[1];
    for (int j = 0; j < 7; j++) {
        ER16(Ct[1], Ct[0], A);
        ER16(B, A, i++);
        ER16(Ct[1], Ct[0], A);
        ER16(C, A, i++);
        ER16(Ct[1], Ct[0], A);
        ER16(D, A, i++);
    }
    ER16(Ct[1], Ct[0], A);

}
void encrypt(uint64_t ct[2], uint64_t const pt[2], uint64_t const K[2]) {
    uint64_t y = pt[0], x = pt[1], b = K[0], a = K[1];
    R(x, y, b);
    for (int i = 0; i < ROUNDS - 1; i++) {
        R(a, b, i);
        R(x, y, b);
    }
    ct[0] = y;
    ct[1] = x;
}
__global__ void speck_exhaustive(uint64_t *ct, uint64_t* pt, uint64_t* K, uint64_t trials ) {    
    uint64_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t b, a, x, y;
//    uint64_t pt0 = pt[0], pt1 = pt[1];
//    uint64_t ct0 = ct[0], ct1 = ct[1];
    uint64_t pt0 = pt[0], pt1 = pt[1];
    uint64_t ct0 = ct[0], ct1 = ct[1];
    for (uint64_t trial = 0; trial < trials; trial++) {        
        b = threadIndex;
        a = trial;
        y = pt0; x = pt1;        
        R(x, y, b);
        for (int i = 0; i < ROUNDS - 1; i++) {
            R(a, b, i);
            R(x, y, b);
        }
        if ((y == ct0) && (x == ct1)) {            K[0] = threadIndex; K[1] = trial;    }
    } 
}
__global__ void speck96_exhaustive(uint32_t* ct, uint32_t* pt, uint32_t* K, uint64_t trials) {
    uint32_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t pt0 = pt[0], pt1 = pt[1], ct0, ct1;
    uint32_t c0 = ct[0], c1 = ct[1];
    uint32_t A, B, C;
    for (uint32_t trial = 0; trial < trials; trial++) {
        uint32_t i = 0;
        ct0 = pt0; ct1 = pt1;
        A = threadIndex;
        B = trial;
        C = 0x13121110l;
#pragma unroll
        for (int j = 0; j < 12; j++) {
            ER32(ct1, ct0, A);
            ER32(B, A, i++);
            ER32(ct1, ct0, A);
            ER32(C, A, i++);
        }
        ER32(ct1, ct0, A);
        ER32(B, A, i++);
        ER32(ct1, ct0, A);
        if ((ct0 == c0) && (ct1 == c1)) { K[0] = threadIndex; K[1] = trial; K[2] = 0x13121110; }
    }
}
__global__ void speck72_exhaustive(uint32_t* ct, uint32_t* pt, uint32_t* K, uint64_t trials) {
    uint32_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t pt0 = pt[0], pt1 = pt[1], ct0, ct1;
    uint32_t c0 = ct[0], c1 = ct[1];
    uint32_t A, B, C;
    for (uint32_t trial = 0; trial < trials; trial++) {
        uint32_t i = 0;
        ct0 = pt0; ct1 = pt1;
        A = threadIndex;
        B = trial;
        C = 0x121110;
#pragma unroll
        for (int j = 0; j < 10; j++) {
            ER24(ct1, ct0, A);
            ER24(B, A, i++);
            ER24(ct1, ct0, A);
            ER24(C, A, i++);
        }
        ER24(ct1, ct0, A);
        ER24(B, A, i++);
        ER24(ct1, ct0, A);
        if ((ct0 == c0) && (ct1 == c1)) { K[0] = threadIndex; K[1] = trial; K[2] = 0x121110; }
    }
}
__global__ void speck64_exhaustive(uint16_t* ct, uint16_t* pt, uint16_t* K, uint32_t trials) {
    uint32_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint16_t pt0 = pt[0], pt1 = pt[1], ct0, ct1;
    uint16_t c0 = ct[0], c1 = ct[1];
    uint16_t A, B, C, D;
    for (uint32_t trial = 0; trial < trials; trial++) {
        uint16_t i = 0;
        ct0 = pt0; ct1 = pt1;
        A = threadIndex>>16;
        B = threadIndex & 0xFFFF;
 //       C = 0x1110;
        C = 0x00ab;
        D = trial;
        for (int j = 0; j < 7; j++) {
            ER16(ct1, ct0, A);
            ER16(B, A, i++);
            ER16(ct1, ct0, A);
            ER16(C, A, i++);
            ER16(ct1, ct0, A);
            ER16(D, A, i++);
        }
        ER16(ct1, ct0, A);
        if ((ct0 == c0) && (ct1 == c1)) { K[0] = threadIndex >> 16; K[1] = threadIndex & 0xffff; K[2] = 0x1110; K[3] = trial;  }
    }
}
__global__ void speck64_exhaustive32bit(uint32_t* ct, uint32_t* pt, uint32_t* K, uint32_t trials) {
    uint32_t threadIndex = (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t pt0 = pt[0], pt1 = pt[1], ct0, ct1;
    uint32_t c0 = ct[0], c1 = ct[1];
    uint32_t A, B, C, D;
    for (uint32_t trial = 0; trial < trials; trial++) {
        uint32_t i = 0;
        ct0 = pt0; ct1 = pt1;
        A = threadIndex >> 16;
        B = threadIndex & 0xFFFF;
        //       C = 0x1110;
        C = 0x00ab;
        D = trial;
        for (int j = 0; j < 7; j++) {
            ER16b(ct1, ct0, A);
            ER16b(B, A, i++);
            ER16b(ct1, ct0, A);
            ER16b(C, A, i++);
            ER16b(ct1, ct0, A);
            ER16b(D, A, i++);
        }
        ER16b(ct1, ct0, A);
        if ((ct0 == c0) && (ct1 == c1)) { K[0] = threadIndex >> 16; K[1] = threadIndex & 0xffff; K[2] = 0x1110; K[3] = trial; }
    }
}

int main_C() {
    uint32_t pt[2] = { 0x736e6165, 0x74614620  };
    uint32_t ct[2] = { 0 }; // Ciphertext: 4175946c 09f7952ec
//    uint32_t K[3] = {   0x03020100, 0x0b0a0908, 0x13121110 };
    uint32_t K[3] = { 0x000015f6, 0x000001ab, 0x13121110 }; 
    //Ciphertext: 0c85aae1 0438f26e5
    uint32_t rk[26] = { 0 };
    Speck6496Encrypt2(pt, ct, K);
    printf("Ciphertext: %08x 0%08x\n", ct[0], ct[1]);
    Speck6496KeySchedule(K, rk);
    for (int i = 0; i < 26; i++) printf("%08x\n",rk[i]);
    Speck6496Encrypt(pt,ct,rk);
    printf("Ciphertext: %08x 0%08x\n", ct[0], ct[1]);
    return 0;
}
//SPECK 96/64
int main96() {
    hipSetDevice(0);
    uint32_t ct[2] = { 0x0c85aae1, 0x0438f26e5 }, pt[2] = { 0x736e6165, 0x74614620 }, K[3] = { 0xffffffff, 0xffffffff, 0xffffffff };
    // corrrect key K[2] = { 0x3, 0x5 };
    uint32_t* ct_d; uint32_t* pt_d; uint32_t* K_d;
    uint32_t trial = 1;
    printf("Trials 2^20 + ");
    scanf_s("%d", &trial);
    trial = (uint32_t)1 << trial;
    // Ciphertext: 9c3df6b05f625cb2 5da73f447979dccd
    // encrypt(ct, pt, K);
    // printf("Ciphertext: %llx %llx\n", ct[0], ct[1]);
    hipMalloc((void**)&ct_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&pt_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&K_d, 3 * sizeof(uint32_t));
    hipMemcpy(pt_d, pt, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(ct_d, ct, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(K_d, K, 3 * sizeof(uint32_t), hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    speck96_exhaustive << <BLOCKS, THREADS >> > (ct_d, pt_d, K_d, trial);
    hipMemcpy(K, K_d, 3 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %08x %08x %08x\n", K[0], K[1], K[2]);
    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
//SPECK 128/128
int main128() { 
    hipSetDevice(0);
    uint64_t ct[2] = { 0x9c3df6b05f625cb2, 0x5da73f447979dccd }, pt[2] = { 0x01234567, 0x89abcdef }, K[2] = { 0xffffffffffffffff, 0xffffffffffffffff };
    // corrrect key K[2] = { 0x3, 0x5 };
    uint64_t *ct_d; uint64_t* pt_d; uint64_t* K_d;
    uint64_t trial = 1;
    printf("Trials 2^20 + ");
    scanf_s("%lld", &trial);
    trial = (uint64_t)1 << trial;
    // Ciphertext: 9c3df6b05f625cb2 5da73f447979dccd
    // encrypt(ct, pt, K);
    // printf("Ciphertext: %llx %llx\n", ct[0], ct[1]);

    hipMalloc((void**)&ct_d, 2 * sizeof(uint64_t));
    hipMalloc((void**)&pt_d, 2 * sizeof(uint64_t));
    hipMalloc((void**)&K_d, 2 * sizeof(uint64_t));
    hipMemcpy(pt_d, pt, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(ct_d, ct, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(K_d, K, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    speck_exhaustive << <BLOCKS, THREADS >> > (ct_d, pt_d, K_d, trial);

    hipMemcpy(K, K_d, 2 * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %llx %llx\n", K[0], K[1]);

    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
//SPECK 64
int main64b() {
// Key: 1918 1110 0908 0100
// Plaintext: 6574 694c
// Ciphertext : a868 42f2
//    uint16_t ct[2] = { 0x42f2, 0xa868 }, pt[2] = {  0x694c, 0x6574 }, K[4] = { 0xffff, 0xffff, 0xffff, 0xffff };
//    uint16_t ct[2] = { 0, 0 }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0x0100, 0x0908, 0x1110, 0x1918 };
//    uint16_t ct[2] = { 0x4ca5, 0xa08c }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0x0001, 0x0098, 0x00ab, 0x00f7 };
    uint16_t ct[2] = { 0x4ca5, 0xa08c }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0xffff, 0xffff, 0xffff, 0xffff };
    uint16_t* ct_d; uint16_t* pt_d; uint16_t* K_d;
    uint32_t trial = 1;
//    Speck6464Encrypt(pt, ct, K);
//    printf("Ciphertext: %04x %04x\n", ct[0], ct[1]);
    printf("Trials 2^20 + ");
    scanf_s("%d", &trial);
    trial = (uint32_t)1 << trial;

    hipMalloc((void**)&ct_d, 2 * sizeof(uint16_t));
    hipMalloc((void**)&pt_d, 2 * sizeof(uint16_t));
    hipMalloc((void**)&K_d, 4 * sizeof(uint16_t));
    hipMemcpy(pt_d, pt, 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(ct_d, ct, 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(K_d, K, 4 * sizeof(uint16_t), hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    speck64_exhaustive << <BLOCKS, THREADS >> > (ct_d, pt_d, K_d, trial);
    hipMemcpy(K, K_d, 4 * sizeof(uint16_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %04x %04x %04x %04x\n", K[0], K[1], K[2], K[3]);
    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
int main64() {
    hipSetDevice(0);
    // Key: 1918 1110 0908 0100
    // Plaintext: 6574 694c
    // Ciphertext : a868 42f2
    //    uint16_t ct[2] = { 0x42f2, 0xa868 }, pt[2] = {  0x694c, 0x6574 }, K[4] = { 0xffff, 0xffff, 0xffff, 0xffff };
    //    uint16_t ct[2] = { 0, 0 }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0x0100, 0x0908, 0x1110, 0x1918 };
    //    uint16_t ct[2] = { 0x4ca5, 0xa08c }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0x0001, 0x0098, 0x00ab, 0x00f7 };
    uint32_t ct[2] = { 0x4ca5, 0xa08c }, pt[2] = { 0x694c, 0x6574 }, K[4] = { 0xffff, 0xffff, 0xffff, 0xffff };
    uint32_t* ct_d; uint32_t* pt_d; uint32_t* K_d;
    uint32_t trial = 1;
    //    Speck6464Encrypt(pt, ct, K);
    //    printf("Ciphertext: %04x %04x\n", ct[0], ct[1]);
    printf("Trials 2^20 + ");
    scanf_s("%d", &trial);
    trial = (uint32_t)1 << trial;

    hipMalloc((void**)&ct_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&pt_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&K_d, 4 * sizeof(uint32_t));
    hipMemcpy(pt_d, pt, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(ct_d, ct, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(K_d, K, 4 * sizeof(uint32_t), hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    speck64_exhaustive32bit << <BLOCKS, THREADS >> > (ct_d, pt_d, K_d, trial);
    hipMemcpy(K, K_d, 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %04x %04x %04x %04x\n", K[0], K[1], K[2], K[3]);
    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
int main72C() {
//Key: 121110 0a0908 020100
//Plaintext : 20796c 6c6172
// Ciphertext : c049a5 385adc
    uint32_t pt[2] = { 0x6c6172, 0x20796c };
    uint32_t ct[2] = { 0 }; // 0x0080d1a9 0x000535548
    uint32_t K[3] = { 0x000100, 0x000908, 0x121110 };   
    Speck6472Encrypt2(pt, ct, K);
    printf("Ciphertext: %08x 0%08x\n", ct[0], ct[1]);
    return 0;
}
int main72() {
    hipSetDevice(0);
    uint32_t ct[2] = { 0x0080d1a9, 0x000535548 }, pt[2] = { 0x6c6172, 0x20796c }, K[3] = { 0xffffffff, 0xffffffff, 0xffffffff };
    // corrrect key K[2] = { 0x3, 0x5 };
    uint32_t* ct_d; uint32_t* pt_d; uint32_t* K_d;
    uint32_t trial = 1;
    printf("Trials 2^20 + ");
    scanf_s("%d", &trial);
    trial = (uint32_t)1 << trial;
    // Ciphertext: 9c3df6b05f625cb2 5da73f447979dccd
    // encrypt(ct, pt, K);
    // printf("Ciphertext: %llx %llx\n", ct[0], ct[1]);
    hipMalloc((void**)&ct_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&pt_d, 2 * sizeof(uint32_t));
    hipMalloc((void**)&K_d, 3 * sizeof(uint32_t));
    hipMemcpy(pt_d, pt, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(ct_d, ct, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(K_d, K, 3 * sizeof(uint32_t), hipMemcpyHostToDevice);
    float time = 0;
    hipEvent_t startx, stopx;
    hipEventCreate(&startx);    hipEventCreate(&stopx);    hipEventRecord(startx);
    speck96_exhaustive << <BLOCKS, THREADS >> > (ct_d, pt_d, K_d, trial);
    hipMemcpy(K, K_d, 3 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipEventRecord(stopx);    hipEventSynchronize(stopx);    hipEventElapsedTime(&time, startx, stopx);
    printf("Captured key: %08x %08x %08x\n", K[0], K[1], K[2]);
    printf("Elapsed time: %f\n", time);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
int main() {
    int choice = 0;
    printf("(1) SPECK-64\n"
        "(2) SPECK-72\n"
        "(3) SPECK-96\n"
        "(4) SPECK-128\n"
        "Choice: "
    );
    scanf_s("%d", &choice);
    if (choice == 1) main64();
    if (choice == 2) main72();
    if (choice == 3) main96();
    if (choice == 4) main128();
}